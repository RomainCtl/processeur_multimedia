#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define MAX_CHAINE 100

#define CALLOC(ptr, nr, type)   if (!(ptr = (type *) calloc((size_t)(nr), sizeof(type)))) { \
                                    printf("Erreur lors de l'allocation memoire \n") ;      \
                                    exit (-1);                                              \
                                }


#define FOPEN(fich,fichier,sens)    if ((fich=fopen(fichier,sens)) == NULL) {                   \
                                        printf("Probleme d'ouverture du fichier %s\n",fichier); \
                                        exit(-1);                                               \
                                    }

#define MIN(a, b)         (a < b ? a : b)
#define MAX(a, b)         (a > b ? a : b)

#define MAX_VALEUR         255
#define MIN_VALEUR         0

#define NBPOINTSPARLIGNES 15

#define false 0
#define true 1
#define boolean int

// Clock
#define initTimer struct timeval tv1, tv2; struct timezone tz
#define startTimer gettimeofday(&tv1, &tz)
#define stopTimer gettimeofday(&tv2, &tz)
#define tpsCalcul ((tv2.tv_sec-tv1.tv_sec)*1000000L + (tv2.tv_usec-tv1.tv_usec)) / 1000


/* KERNEL CUDA */
__global__ void add_vec_scalaire_gpu(int **image, int **res, long N, int x, int le_min, float etalement) {
    long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
    int j;
    if (i < N) {
        for (j = 0 ; j < x ; j++) {
            res[i][j] = ((image[i][j] - le_min) * etalement);
        }
    }
}


int main(argc, argv) int argc; char* argv[]; {
    /*========================================================================*/
    /* Declaration de variables et allocation memoire */
    /*========================================================================*/

    int i, j, n;

    int LE_MIN = MAX_VALEUR;
    int LE_MAX = MIN_VALEUR;

    float ETALEMENT = 0.0;

    int** image;
    int** resultat;
    int X, Y, x, y;
    int TailleImage;

    int NbResultats, quelle_ligne, lignes;
    int* la_ligne;

    int P;

    FILE* Src, * Dst;

    char SrcFile[MAX_CHAINE];
    char DstFile[MAX_CHAINE];

    char ligne[MAX_CHAINE];

    boolean inverse = false;

    char *Chemin;

    initTimer; //

    /*========================================================================*/
    /* Recuperation des parametres                                                */
    /*========================================================================*/

    sscanf(argv[1], "%s", SrcFile);

    sprintf(DstFile, "%s.new", SrcFile);

    /*========================================================================*/
    /* Recuperation de l'endroit ou l'on travail                                */
    /*========================================================================*/

    CALLOC(Chemin, MAX_CHAINE, char);
    Chemin = getenv("PWD");
    printf("Repertoire de travail : %s \n\n", Chemin);

    /*========================================================================*/
    /* Ouverture des fichiers                                                */
    /*========================================================================*/

    printf("Operations sur les fichiers\n");

    FOPEN(Src, SrcFile, "r");
    printf("\t Fichier source ouvert (%s) \n", SrcFile);

    FOPEN(Dst, DstFile, "w");
    printf("\t Fichier destination ouvert (%s) \n", DstFile);

    /*========================================================================*/
    /* On effectue la lecture du fichier source */
    /*========================================================================*/

    printf("\t Lecture entete du fichier source ");

    for (i = 0; i < 2; i++) {
        fgets(ligne, MAX_CHAINE, Src);
        fprintf(Dst, "%s", ligne);
    }

    fscanf(Src, " %d %d\n", &X, &Y);
    fprintf(Dst, " %d %d\n", X, Y);

    fgets(ligne, MAX_CHAINE, Src);        /* Lecture du 255         */
    fprintf(Dst, "%s", ligne);

    printf(": OK \n");

    /*========================================================================*/
    /* Allocation memoire pour l'image source et l'image resultat                 */
    /*========================================================================*/

    CALLOC(image, Y + 1, int*);
    CALLOC(resultat, Y + 1, int*);
    for (i = 0;i < Y;i++) {
        CALLOC(image[i], X + 1, int);
        CALLOC(resultat[i], X + 1, int);
        for (j = 0;j < X;j++) {
            image[i][j] = 0;
            resultat[i][j] = 0;
        }
    }
    printf("\t\t Initialisation de l'image [%d ; %d] : Ok \n", X, Y);

    TailleImage = X * Y;

    x = 0;
    y = 0;

    lignes = 0;

    /*========================================================================*/
    /* Lecture du fichier pour remplir l'image source                         */
    /*========================================================================*/

    while (!feof(Src)) {
        n = fscanf(Src, "%d", &P);
        image[y][x] = P;
        LE_MIN = MIN(LE_MIN, P);
        LE_MAX = MAX(LE_MAX, P);
        x++;
        if (n == EOF || (x == X && y == Y - 1)) {
            break;
        }
        if (x == X) {
            x = 0;
            y++;
        }
    }
    fclose(Src);
    printf("\t Lecture du fichier image : Ok \n\n");

    /*========================================================================*/
    /* Calcul du facteur d'etalement                                        */
    /*========================================================================*/

    if (inverse) {
        ETALEMENT = 0.2;
    }
    else {
        ETALEMENT = (float)(MAX_VALEUR - MIN_VALEUR) / (float)(LE_MAX - LE_MIN);
    }

    /*========================================================================*/
    /* Calcul de chaque nouvelle valeur de pixel                              */
    /*========================================================================*/

    int tailleVecteur = Y;
    long blocksize = 1

    int **cudaVec;
    int **cudaRes;

    // Select cuda GPU device to use (if multiple device)
    hipSetDevice(0); // TODO make it as args ?

    if (hipMalloc((void **)&cudaVec, TailleImage) == hipErrorOutOfMemory) {
        printf("Allocation memoire qui pose probleme (cudaVec) \n");
    }
    if (hipMalloc((void **)&cudaRes, TailleImage)  == hipErrorOutOfMemory) {
        printf("Allocation memoire qui pose probleme (cudaRes) \n");
    }

    long dimBlock = blocksize;
    long dimGrid = tailleVecteur/blocksize;
    if ((tailleVecteur % blocksize) != 0) {
        dimGrid++;
    }

    int res = hipMemcpy(&cudaVec[0], image[0], TailleImage * sizeof(int), hipMemcpyHostToDevice);

    int res = hipMemcpy(&cudaVec[0], &image[0], TailleImage * sizeof(int), hipMemcpyHostToDevice);

    printf("Copy CPU -> GPU %d \n",res);

    startTimer;
    add_vec_scalaire_gpu<<<dimGrid, dimBlock>>>(cudaVec, cudaRes, Y, X, LE_MIN, ETALEMENT);
    stopTimer;

    hipMemcpy(&resultat[0], &cudaRes[0], TailleImage * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(cudaVec);
    hipFree(cudaRes);

    /*========================================================================*/
    /* Sauvegarde de l'image dans le fichier resultat                         */
    /*========================================================================*/

    n = 0;
    for (i = 0; i < Y; i++) {
        for (j = 0; j < X; j++) {

            fprintf(Dst, "%3d ", resultat[i][j]);
            n++;
            if (n == NBPOINTSPARLIGNES) {
                n = 0;
                fprintf(Dst, "\n");
            }
        }
    }

    fprintf(Dst, "\n");
    fclose(Dst);

    printf("\n");

    /*========================================================================*/
    /* Fin du programme principal        */
    /*========================================================================*/

    exit(0);
}
