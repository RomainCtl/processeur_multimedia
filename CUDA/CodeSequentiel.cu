
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define MAX_CHAINE 100

#define CALLOC(ptr, nr, type)   if (!(ptr = (type *) calloc((size_t)(nr), sizeof(type)))) { \
                                    printf("Erreur lors de l'allocation memoire \n") ;      \
                                    exit (-1);                                              \
                                }


#define FOPEN(fich,fichier,sens)    if ((fich=fopen(fichier,sens)) == NULL) {                   \
                                        printf("Probleme d'ouverture du fichier %s\n",fichier); \
                                        exit(-1);                                               \
                                    }

#define MIN(a, b)         (a < b ? a : b)
#define MAX(a, b)         (a > b ? a : b)

#define MAX_VALEUR         255
#define MIN_VALEUR         0

#define NBPOINTSPARLIGNES 15

#define MAX_DIM_GRID 65535
#define MAX_DIM_BLOCK 1024

#define false 0
#define true 1
#define boolean int

// Clock
#define initTimer struct timeval tv1, tv2; struct timezone tz
#define startTimer gettimeofday(&tv1, &tz)
#define stopTimer gettimeofday(&tv2, &tz)
#define tpsCalcul ((tv2.tv_sec-tv1.tv_sec)*1000000L + (tv2.tv_usec-tv1.tv_usec))


/* KERNEL CUDA */
__global__ void add_vec_scalaire_gpu(int *image, int *res, long N, int le_min, float etalement) {
    long i = (long)blockIdx.x * (long)blockDim.x + (long)threadIdx.x;
    if (i < N) {
        res[i] = ((image[i] - le_min) * etalement);
    }
}


int main(int argc, char *argv[]) {
    /*========================================================================*/
    /* Declaration de variables et allocation memoire */
    /*========================================================================*/

    if (argc < 2) {
        printf("Usage: ./CodeSequentiel <path_to_image> [<blocksize>]\n");
        exit(0);
    }

    long blocksize = 1; // default value
    if (argc == 3) {
        blocksize = atoi(argv[2]);
    }

    int i, n;

    int LE_MIN = MAX_VALEUR;
    int LE_MAX = MIN_VALEUR;

    float ETALEMENT = 0.0;

    int* image;
    int* resultat;
    int X, Y, x, y;
    int TailleImage;

    int P;

    FILE* Src, * Dst;

    char SrcFile[MAX_CHAINE];
    char DstFile[MAX_CHAINE];

    char ligne[MAX_CHAINE];

    boolean inverse = false;

    char *Chemin;

    initTimer; //

    /*========================================================================*/
    /* Recuperation des parametres                                                */
    /*========================================================================*/

    sscanf(argv[1], "%s", SrcFile);

    sprintf(DstFile, "%s.new", SrcFile);

    /*========================================================================*/
    /* Recuperation de l'endroit ou l'on travail                                */
    /*========================================================================*/

    CALLOC(Chemin, MAX_CHAINE, char);
    Chemin = getenv("PWD");
    printf("Repertoire de travail : %s \n\n", Chemin);

    /*========================================================================*/
    /* Ouverture des fichiers                                                */
    /*========================================================================*/

    printf("Operations sur les fichiers\n");

    FOPEN(Src, SrcFile, "r");
    printf("\t Fichier source ouvert (%s) \n", SrcFile);

    FOPEN(Dst, DstFile, "w");
    printf("\t Fichier destination ouvert (%s) \n", DstFile);

    /*========================================================================*/
    /* On effectue la lecture du fichier source */
    /*========================================================================*/

    printf("\t Lecture entete du fichier source ");

    for (i = 0; i < 2; i++) {
        fgets(ligne, MAX_CHAINE, Src);
        fprintf(Dst, "%s", ligne);
    }

    fscanf(Src, " %d %d\n", &X, &Y);
    fprintf(Dst, " %d %d\n", X, Y);

    fgets(ligne, MAX_CHAINE, Src);        /* Lecture du 255         */
    fprintf(Dst, "%s", ligne);

    printf(": OK \n");

    /*========================================================================*/
    /* Allocation memoire pour l'image source et l'image resultat                 */
    /*========================================================================*/

    TailleImage = X * Y;

    CALLOC(image, TailleImage, int);
    CALLOC(resultat, TailleImage, int);
    for (i = 0;i < TailleImage;i++) {
        image[i] = 0;
        resultat[i] = 0;
    }

    x = 0;
    y = 0;

    printf("\t\t Initialisation de l'image [%d ; %d] : Ok \n", X, Y);

    /*========================================================================*/
    /* Lecture du fichier pour remplir l'image source                         */
    /*========================================================================*/

    while (!feof(Src)) {
        n = fscanf(Src, "%d", &P);
        image[y+x] = P;
        LE_MIN = MIN(LE_MIN, P);
        LE_MAX = MAX(LE_MAX, P);
        x++;
        if (n == EOF || (x == X && y == Y - 1)) {
            break;
        }
        if (x == X) {
            x = 0;
            y++;
        }
    }
    fclose(Src);
    printf("\t Lecture du fichier image : Ok \n\n");

    /*========================================================================*/
    /* Calcul du facteur d'etalement                                        */
    /*========================================================================*/

    if (inverse) {
        ETALEMENT = 0.2;
    }
    else {
        ETALEMENT = (float)(MAX_VALEUR - MIN_VALEUR) / (float)(LE_MAX - LE_MIN);
    }

    /*========================================================================*/
    /* Calcul de chaque nouvelle valeur de pixel                              */
    /*========================================================================*/

    int tailleVecteur = TailleImage;
    //long blocksize = 1; // TODO can change from args

    long size = sizeof(int)*tailleVecteur;

    int *cudaVec;
    int *cudaRes;

    // Select cuda GPU device to use (if multiple device)
    hipSetDevice(0);

    if (hipMalloc((void **)&cudaVec, size) == hipErrorOutOfMemory) {
        printf("Allocation memoire qui pose probleme (cudaVec) \n");
    }
    if (hipMalloc((void **)&cudaRes, size)  == hipErrorOutOfMemory) {
        printf("Allocation memoire qui pose probleme (cudaRes) \n");
    }

    long dimBlock = blocksize;
    long dimGrid = tailleVecteur/blocksize;
    if ((tailleVecteur % blocksize) != 0) {
        dimGrid++;
    }

    int res = hipMemcpy(&cudaVec[0], &image[0], size, hipMemcpyHostToDevice);

    printf("Copy CPU -> GPU %d \n",res);
    printf("dimBlock: %ld | dimGrid: %ld\n", dimBlock, dimGrid);

    startTimer;
    add_vec_scalaire_gpu<<<dimGrid, dimBlock>>>(cudaVec, cudaRes, tailleVecteur, LE_MIN, ETALEMENT);
    hipDeviceSynchronize();
    stopTimer;

    printf("Duration %ld", tpsCalcul);

    hipMemcpy(&resultat[0], &cudaRes[0], size, hipMemcpyDeviceToHost);

    hipFree(cudaVec);
    hipFree(cudaRes);

    /*========================================================================*/
    /* Sauvegarde de l'image dans le fichier resultat                         */
    /*========================================================================*/

    n = 0;
    for (i = 0; i < TailleImage ; i++) {
        fprintf(Dst, "%3d ", resultat[i]);
        n++;
        if (n == NBPOINTSPARLIGNES) {
            n = 0;
            fprintf(Dst, "\n");
        }
    }

    fprintf(Dst, "\n");
    fclose(Dst);

    printf("\n");

    /*========================================================================*/
    /* Fin du programme principal        */
    /*========================================================================*/

    exit(0);
}
